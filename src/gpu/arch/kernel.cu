#include "hip/hip_runtime.h"
#include "kernel.hpp"

__global__
void compute_kernel(float *vec_a, float *vec_b, float *vec_c, int n) {
  float a = vec_a[0];
  float b = vec_b[0];
  float c = 0.0;

  #pragma unroll 2000
  for(long i=0; i < BENCHMARK_GPU_ITERS; i++) {
    c = (c * a) + b;
  }

  vec_c[0] = c;
}
