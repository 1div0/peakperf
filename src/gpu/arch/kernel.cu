#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>

#include "maxwell.hpp"

// Avoid ArchLinux package warning "WARNING: Package contains reference to $srcdir"
#define __FILENAME__ "kernel.cu"

enum {
  ARCH_MAXWELL,
  ARCH_UNKNOWN
};

static const char *uarch_str[] = {
  /*[ARCH_MAXWELL]    = */ "Maxwell",
};

struct benchmark_gpu {
  int nbk; // Blocks per thread
  int tpb; // Threads per block
  int n;
  double gflops;
  void(*compute_function)(float *, float *, float *, int);
  float *d_A;
  float *d_B;
  float *d_C;
};

// We assume only one gpu is present...
struct gpu {
  int compute_capability;
  int sm_count;
  char uarch;
  char* name;
};

struct gpu* get_gpu_info() {
  struct gpu* gpu = (struct gpu *) malloc(sizeof(struct gpu));

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);

  int gpu_name_len = strlen(deviceProp.name);
  gpu->compute_capability = deviceProp.major * 10 + deviceProp.minor;
  gpu->sm_count = deviceProp.multiProcessorCount;
  gpu->name = (char *) malloc(sizeof(char) * (gpu_name_len + 1));
  memset(gpu->name, 0, gpu_name_len + 1);
  strncpy(gpu->name, deviceProp.name, gpu_name_len);

  switch(gpu->compute_capability) {
    case 52:
      gpu->uarch = ARCH_MAXWELL;
      break;
    default:
      printf("Invalid uarch found: %d.%d\n", deviceProp.major, deviceProp.minor);
      return NULL;
  }

  return gpu;
}

struct benchmark_gpu* init_benchmark_gpu(struct gpu* gpu, int nbk, int tpb) {
  struct benchmark_gpu* bench = (struct benchmark_gpu *) malloc(sizeof(struct benchmark_gpu));

  // TODO: Dont ignore nbk, tpb
  bench->nbk = gpu->sm_count;
  bench->tpb = 1024;
  bench->n = gpu->sm_count * bench->tpb;

  switch(gpu->uarch) {
    case ARCH_MAXWELL:
      bench->compute_function = matrixMul_maxwell;
      bench->gflops = (double)(KERNEL_ITERS * 2 * (long)bench->n * WORK_MAXWELL)/(long)1000000000;
      break;
    default:
      return NULL;
  }

  hipError_t err = hipSuccess;
  float *h_A;
  float *h_B;
  int size = bench->n * sizeof(float);

  if ((err = hipHostMalloc((void **)&h_A, size)) != hipSuccess) {
    printf("[%s:%d]%s: %s\n", __FILENAME__, __LINE__, hipGetErrorName(err), hipGetErrorString(err));
    return NULL;
  }

  if ((err = hipHostMalloc((void **)&h_B, size)) != hipSuccess) {
    printf("[%s:%d]%s: %s\n", __FILENAME__, __LINE__, hipGetErrorName(err), hipGetErrorString(err));
    return NULL;
  }

  for (int i = 0; i < bench->n; i++) {
    h_A[i] = rand()/(float)RAND_MAX;
    h_B[i] = rand()/(float)RAND_MAX;
  }

  if ((err = hipMalloc((void **) &(bench->d_A), size)) != hipSuccess) {
    printf("[%s:%d]%s: %s\n", __FILENAME__, __LINE__, hipGetErrorName(err), hipGetErrorString(err));
    return NULL;
  }

  if ((err = hipMalloc((void **) &(bench->d_B), size)) != hipSuccess) {
    printf("[%s:%d]%s: %s\n", __FILENAME__, __LINE__, hipGetErrorName(err), hipGetErrorString(err));
    return NULL;
  }

  if ((err = hipMalloc((void **) &(bench->d_C), size)) != hipSuccess) {
    printf("[%s:%d]%s: %s\n", __FILENAME__, __LINE__, hipGetErrorName(err), hipGetErrorString(err));
    return NULL;
  }

  if ((err = hipMemcpy(bench->d_A, h_A, size, hipMemcpyHostToDevice)) != hipSuccess) {
    printf("[%s:%d]%s: %s\n", __FILENAME__, __LINE__, hipGetErrorName(err), hipGetErrorString(err));
    return NULL;
  }

  if ((err = hipMemcpy(bench->d_B, h_B, size, hipMemcpyHostToDevice)) != hipSuccess) {
    printf("[%s:%d]%s: %s\n", __FILENAME__, __LINE__, hipGetErrorName(err), hipGetErrorString(err));
    return NULL;
  }

  return bench;
}

const char* get_benchmark_name_gpu(struct benchmark_gpu* bench) {
  return uarch_str[0];
}

double get_gflops_gpu(struct benchmark_gpu* bench) {
  return bench->gflops;
}

bool compute_gpu(struct benchmark_gpu* bench) {
  hipError_t err = hipSuccess;
  dim3 dimGrid(bench->nbk, 1, 1);
  dim3 dimBlock(bench->tpb, 1, 1);

  bench->compute_function<<<dimGrid, dimBlock>>>(bench->d_A, bench->d_B, bench->d_C, bench->n);

  hipDeviceSynchronize();

  if ((err = hipGetLastError()) != hipSuccess) {
    printf("[%s:%d]%s: %s\n", __FILENAME__, __LINE__, hipGetErrorName(err), hipGetErrorString(err));
    return false;
  }
  return true;
}

void exit_benchmark_gpu() {
  hipDeviceReset();
}

char* get_str_gpu_name(struct gpu* gpu) {
  return gpu->name;
}

const char* get_str_gpu_uarch(struct gpu* gpu) {
  return uarch_str[gpu->uarch];
}

int get_n_blocks(struct benchmark_gpu* bench) {
  return bench->nbk;
}

int get_threads_per_block(struct benchmark_gpu* bench) {
  return bench->tpb;
}

